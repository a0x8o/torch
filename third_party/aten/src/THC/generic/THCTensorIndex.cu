
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorIndex.cu"
#else

void THCTensor_(indexCopy_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexCopy)(state, dst, dim, indices_, src);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexCopy)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, src);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) == 1, 3,
             "expecting vector of indices");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");
  THArgCheck(numIndices == src->size[dim], 4, "length of src.size[dim] is not equal to length of indices");

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstCopyDimSize = THCTensor_(size)(state, dst, dim);
  ptrdiff_t sliceSize = srcTotalSize / numIndices;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexCopySmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>       \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(           \
      dstInfo, srcInfo, indicesInfo,                            \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexCopyLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>       \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(           \
      dstInfo, srcInfo, indicesInfo,                            \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        LARGE_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        LARGE_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(take)(THCState *state, THCTensor *dst, THCTensor *src, THCudaLongTensor *index)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  THCTensor_(resizeNd)(state, dst, index->nDimension, index->size, NULL);

  dispatchTakePut<real, TensorTakeOp>(state, src, dst, index);
}

static void THCTensor_(sort_indices)(THCState *state, THCudaLongTensor *index, THCTensor *src) {
  THCThrustAllocator thrustAlloc(state);

  auto index_iter = thrust::device_ptr<int64_t>(THCudaLongTensor_data(state, index));
  auto src_iter = thrust::device_ptr<real>(THCTensor_(data)(state, src));
  auto numel = THCTensor_(numel)(state, src);

  thrust::sort_by_key(
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
    index_iter, index_iter + numel,
    src_iter, ThrustLTOp<int64_t>());
}

void THCTensor_(put)(THCState *state, THCTensor *dst, THCudaLongTensor *index, THCTensor *src, int accumulate)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  ptrdiff_t dstSize = THCTensor_(nElement)(state, dst);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, index);
  THArgCheck(THCTensor_(nElement)(state, src) == numIndices,
    3, "src should have the same number of elements as index");

  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  int srcDims = THCTensor_(nDimension)(state, src);
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");

  if (accumulate) {
    // wrap indices so to replace negative indices
    THCudaLongTensor* sorted_index = THCudaLongTensor_new(state);
    THCudaLongTensor_resizeAs(state, sorted_index, index);
    THC_pointwiseApply2(state, sorted_index, index, WrapIndexOp(dstSize));

    THCTensor* sorted_src = THCTensor_(newClone)(state, src);

    THCTensor_(sort_indices)(state, sorted_index, sorted_src);
    dispatchTakePut<real, TensorPutAccumulateOp>(state, dst, sorted_src, sorted_index);

    THCTensor_(free)(state, sorted_src);
    THCudaLongTensor_free(state, sorted_index);
  } else {
    dispatchTakePut<real, TensorPutOp>(state, dst, src, index);
  }
}

void THCTensor_(indexAdd_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexAdd)(state, dst, dim, indices_, src);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexAdd)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, src);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) == 1, 3,
             "expecting vector of indices");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");
  THArgCheck(numIndices == src->size[dim], 4, "length of src.size[dim] is not equal to length of indices");

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstAddDimSize = THCTensor_(size)(state, dst, dim);
  ptrdiff_t sliceSize = srcTotalSize / numIndices;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexAddSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(   \
      dstInfo, srcInfo, indicesInfo,                    \
      dstAddDim, srcAddDim, sliceSize, dstAddDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexAddLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(   \
      dstInfo, srcInfo, indicesInfo,                    \
      dstAddDim, srcAddDim, sliceSize, dstAddDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        LARGE_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        LARGE_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(indexFill_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, real val)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexFill)(state, dst, dim, indices_, val);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexFill)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, real val)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));
  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, dst);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) == 1, 3,
             "expecting vector of indices");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t dstFillDimSize = THCTensor_(size)(state, dst, dim);
  ptrdiff_t sliceSize = dstTotalSize / dstFillDimSize;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)  \
  indexFillSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(   \
      dstInfo, indicesInfo,                             \
      dstFillDim, sliceSize, dstFillDimSize, val);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)  \
  indexFillLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(   \
      dstInfo, indicesInfo,                             \
      dstFillDim, sliceSize, dstFillDimSize, val);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, -2);
      } else if (dstInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, -2);
      } else if (dstInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1);
      }
    } else {
      if (dstInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, -2);
      } else if (dstInfo.dims == 2 && indContig) {
        LARGE_INDEX(real, unsigned int, 2, -2);
      } else if (dstInfo.dims == 3 && indContig) {
        LARGE_INDEX(real, unsigned int, 3, -2);
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}


void THCTensor_(indexSelect_long)(THCState *state, THCTensor *dst, THCTensor *src, int dim, THLongTensor *indices)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THArgCheck(indices->nDimension <= 1, 3, "Index is supposed to be an empty tensor or a vector");

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexSelect)(state, dst, src, dim, indices_);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexSelect)(THCState *state, THCTensor *dst, THCTensor *src, int dim, THCudaLongTensor *indices)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, dst, src, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, src);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) <= 1, 3,
             "Index is supposed to be an empty tensor or a vector");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");

  THLongStorage *newSize;

  if (numIndices == 0) {
    newSize = THCTensor_(newSizeOf)(state, src);
    THLongStorage_set(newSize, 0, numIndices);
    THCTensor_(resize)(state, dst, newSize, NULL);
    THLongStorage_free(newSize);
    return;
  }

  newSize = THCTensor_(newSizeOf)(state, src);
  THLongStorage_set(newSize, dim, numIndices);
  THCTensor_(resize)(state, dst, newSize, NULL);
  THLongStorage_free(newSize);

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t srcSelectDimSize = THCTensor_(size)(state, src, dim);
  ptrdiff_t sliceSize = dstTotalSize / numIndices;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexSelectSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(           \
      dstInfo, srcInfo, indicesInfo,                            \
      dstSelectDim, srcSelectDim, sliceSize, srcSelectDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)         \
  indexSelectLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                   \
      dstInfo, srcInfo, indicesInfo,                                    \
      dstSelectDim, srcSelectDim, dstTotalSize, sliceSize, srcSelectDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        LARGE_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        LARGE_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

#define MAX_ADVINDEX_CALC_DIMS 5

void THCTensor_(calculateAdvancedIndexingOffsets)(
  THCState *state,
  THCudaLongTensor *output,
  THCTensor *indexed,
  ptrdiff_t baseOffset,
  THCudaLongTensor **indexers)
{
  int ndim = THCTensor_(nDimension)(state, indexed);
  THAssert(ndim <= MAX_ADVINDEX_CALC_DIMS);

  // Assert all Tensors are on the same GPU, and that the indexing Tensors are
  // contiguous
  for (int i = 0; i < ndim; ++i) {
    if (indexers[i] != NULL) {
      THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, output, indexers[i]));
      THAssert(THCudaLongTensor_isContiguous(state, indexers[i]));
    }
  }

  // Set grid, block dims
  ptrdiff_t nElement = THCudaLongTensor_nElement(state, output);
  const dim3 block = getApplyBlock();
  dim3 grid;
  THAssert(getApplyGrid(state, nElement, grid));

#define HANDLE_CASE(INDEX_TYPE, DIMS)                                                           \
{                                                                                               \
  LinearIndexCalcData<INDEX_TYPE, DIMS> data;                                                   \
  for (int i = 0; i < DIMS; ++i) {                                                              \
    data.baseSizes[i] = THCTensor_(size)(state, indexed, i);                                    \
    data.sizes[i] = indexers[i] != NULL ?                                                       \
      THCudaLongTensor_nElement(state, indexers[i]) :                                           \
        THCTensor_(size)(state, indexed, i);                                                    \
    data.strides[i] = THCTensor_(stride)(state, indexed, i);                                    \
    data.advIndexTensors[i] = indexers[i] != NULL ?                                             \
      THCudaLongTensor_data(state, indexers[i]) : NULL;                                         \
  }                                                                                             \
                                                                                                \
  calculateLinearIndices<INDEX_TYPE, DIMS>                                                      \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(                                     \
    THCudaLongTensor_data(state, output),                                                       \
    nElement,                                                                                   \
    baseOffset,                                                                                 \
    data                                                                                        \
  );                                                                                            \
}

#define RUN_T(INDEX_TYPE)         \
  switch (ndim) {                 \
    case 1:                       \
      HANDLE_CASE(INDEX_TYPE, 1)  \
      break;                      \
    case 2:                       \
      HANDLE_CASE(INDEX_TYPE, 2)  \
      break;                      \
    case 3:                       \
      HANDLE_CASE(INDEX_TYPE, 3)  \
      break;                      \
    case 4:                       \
      HANDLE_CASE(INDEX_TYPE, 4)  \
      break;                      \
    case 5:                       \
      HANDLE_CASE(INDEX_TYPE, 5)  \
      break;                      \
    default:                      \
      THAssert(false);            \
}

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, indexed)) {
    RUN_T(unsigned int);
  } else {
    RUN_T(uint64_t);
  }

#undef HANDLE_CASE
#undef RUN_T

  THCudaCheck(cudaGetLastError());
}

#undef MAX_ADVINDEX_CALC_DIMS

#endif

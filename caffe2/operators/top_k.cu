#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/operators/top_k.h"

#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include "caffe2/core/context.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/top_k_heap_selection.cuh"
#include "caffe2/operators/top_k_radix_selection.cuh"

namespace caffe2 {

// Converts a matrix of size [outerSize, k] containing
// row-wise indices into global (linearized) indices from an original
// matrix of [outerSize, innerSize]
template <typename Index>
__global__ void linearizeRowIndices(
    Index* in,
    Index* out,
    int outerSize,
    int innerSize,
    int k) {
  if (blockIdx.x < outerSize) {
    in += (Index)blockIdx.x * k;
    out += (Index)blockIdx.x * k;

    auto indexOffset = (Index)blockIdx.x * innerSize;
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
      out[i] = in[i] + indexOffset;
    }
  }
}

template <>
class TopKOp<float, HIPContext> : public Operator<HIPContext> {
 public:
  TopKOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws),
        OP_SINGLE_ARG(int, "k", k_, -1) {}

  bool RunOnDevice() override;

 private:
  int k_;
};

bool TopKOp<float, HIPContext>::RunOnDevice() {
  auto& input = Input(0);
  auto* values = Output(0);
  auto* indices = Output(1);
  auto* flatten_indices = OutputSize() > 2 ? Output(2) : nullptr;

  vector<TIndex> in_dims = input.dims();
  CAFFE_ENFORCE(
      in_dims.back() >= k_, "k argment should not be greater than last dim");

  vector<TIndex> out_dims = in_dims;
  out_dims.back() = k_;

  // Get the batch size
  size_t outerSize = 1;
  for (int i = 0; i < in_dims.size() - 1; ++i) {
    outerSize *= in_dims[i];
  }

  values->Resize(out_dims);
  indices->Resize(out_dims);
  if (flatten_indices) {
    flatten_indices->Resize(outerSize * k_);
  }

  // Right now, the top-k operator only supports max-k
  constexpr bool kDir = true;

  if (k_ <= 512) {
    // heap selection is possible
    constexpr int kBlockSize = 256;
    int numWarps = kBlockSize / kWarpSize;

    auto grid = outerSize;
    auto block = kBlockSize;

#define RUN_HEAP(HEAP_SIZE)                                               \
  do {                                                                    \
    int smem = numWarps * HEAP_SIZE * (sizeof(float) + sizeof(TIndex));   \
                                                                          \
    selectRowsViaHeap<float, TIndex, TIndex, kBlockSize, HEAP_SIZE, kDir> \
        <<<grid, block, smem, context_.hip_stream()>>>(                  \
            input.data<float>(),                                          \
            values->mutable_data<float>(),                                \
            indices->mutable_data<TIndex>(),                              \
            kDir ? -std::numeric_limits<float>::infinity()                \
                 : std::numeric_limits<float>::infinity(),                \
            kDir ? -std::numeric_limits<TIndex>::max()                    \
                 : std::numeric_limits<float>::max(),                     \
            outerSize,                                                    \
            in_dims.back(),                                               \
            k_);                                                          \
  } while (false)

    if (k_ <= 32) {
      RUN_HEAP(32);
    } else if (k_ <= 128) {
      RUN_HEAP(128);
    } else {
      RUN_HEAP(512);
    }

#undef RUN_HEAP

  } else {
    // k is too large, use radix selection instead
    auto grid = outerSize;
    auto block = std::min(
        math::roundUp((int)in_dims.back(), kWarpSize), CAFFE_CUDA_NUM_THREADS);

    // Radix selection required
    gatherTopK<float, kDir, TIndex><<<grid, block, 0, context_.hip_stream()>>>(
        input.data<float>(),
        in_dims.back(),
        k_,
        outerSize,
        values->mutable_data<float>(),
        indices->mutable_data<TIndex>());

    // Unfortunately the output is not currently sorted, and there is
    // no batch sorting utility available. Iterate over all of the
    // slices and sort them in-place using Thrust.
    for (int slice = 0; slice < outerSize; ++slice) {
      thrust::sort_by_key(
          thrust::cuda::par.on(context_.hip_stream()),
          values->mutable_data<float>() + slice * k_,
          values->mutable_data<float>() + slice * k_ + k_,
          indices->mutable_data<TIndex>() + slice * k_,
          thrust::greater<float>());
    }
  }

  // Now that we've completed writing the indices, linearize the
  // indices if we need it
  if (flatten_indices) {
    linearizeRowIndices<TIndex>
        <<<outerSize, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            indices->mutable_data<TIndex>(),
            flatten_indices->mutable_data<TIndex>(),
            outerSize,
            in_dims.back(),
            k_);
  }

  return true;
}

REGISTER_HIP_OPERATOR(TopK, TopKOp<float, HIPContext>);

__global__ void fillValuesWithIndicesKernel(
    const float* values,
    const TIndex* indices,
    const TIndex k,
    const TIndex orignal_last_dim,
    const TIndex length,
    float* output) {
  CUDA_1D_KERNEL_LOOP(i, length) {
    int first_dim = i / k;
    int idx = orignal_last_dim * first_dim + indices[i];
    output[idx] = values[i];
  }
}

template <>
bool TopKGradientOp<float, HIPContext>::RunOnDevice() {
  auto& values = Input(0);
  auto& indices = Input(1);
  auto& original_input = Input(2);

  vector<TIndex> in_dims = values.dims();

  // Linearize input tensor except for last dimension
  // e.g. [3, 4, 5] -> [12, 5]
  // [5] -> [5]
  TIndex flatten_shape[] = {size_to_dim_(in_dims.size() - 1, in_dims),
                            in_dims[in_dims.size() - 1]};

  vector<TIndex> original_dims = original_input.dims();
  auto* output = Output(0);
  output->Resize(original_dims);

  float* output_data = output->mutable_data<float>();
  math::Set<float, HIPContext>(
      output->size(), float(0), output_data, &context_);

  int length = flatten_shape[0] * flatten_shape[1];
  if (length == 0) { // for empty batch
    return true;
  }

  int num_threads = std::min(CAFFE_CUDA_NUM_THREADS, length);
  int blocks = math::divUp(length, num_threads);

  fillValuesWithIndicesKernel<<<
      blocks,
      num_threads,
      0,
      context_.hip_stream()>>>(
      values.data<float>(),
      indices.data<TIndex>(),
      flatten_shape[1],
      original_dims.back(),
      length,
      output_data);

  return true;
}

REGISTER_HIP_OPERATOR(TopKGradient, TopKGradientOp<float, HIPContext>);
} // namespace caffe2

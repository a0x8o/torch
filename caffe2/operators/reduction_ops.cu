#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/reduction_ops.h"

namespace caffe2 {
namespace {

REGISTER_HIP_OPERATOR(SumElements, SumElementsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SumSqrElements, SumSqrElementsOp<float, HIPContext>);

REGISTER_HIP_OPERATOR(
    SumElementsGradient,
    SumElementsGradientOp<float, HIPContext>);

template <typename T>
__global__ void
SumElementsGradientKernel(bool average, const int N, const T* dY, T* dX) {
  const T value = average ? (*dY) / N : *dY;
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = value;
  }
}
} // namespace

template <>
bool SumElementsGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& dY = Input(1);
  DCHECK_EQ(dY.size(), 1);
  auto* dX = Output(0);
  dX->ResizeLike(X);
  SumElementsGradientKernel<float><<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      average_, X.size(), dY.data<float>(), dX->mutable_data<float>());
  return true;
}

} // namespace caffe2

#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/math_ops.h"

namespace caffe2 {

struct SqrCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    math::Sqr<T, HIPContext>(n, x, y, device_context);
  }
};

template <typename T>
__global__ void SignKernel(int n, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = (-T(1) * (x[i] < 0)) + (x[i] > 0);
  }
}

struct SignCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    SignKernel<<<
        CAFFE_GET_BLOCKS(n),
        CAFFE_CUDA_NUM_THREADS,
        0,
        device_context->hip_stream()>>>(n, x, y);
  }
};

REGISTER_HIP_OPERATOR(
    Sqr,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SqrCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    Sign,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SignCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    Pow,
    UnaryElementwiseWithArgsOp<TensorTypes<float>, HIPContext, PowFunctor>);
}

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/math_ops.h"

namespace caffe2 {

struct SqrCUDAFunctor {
  template <typename T>
  inline void
  operator()(const int n, const T* x, T* y, HIPContext* device_context) {
    math::Sqr<T, HIPContext>(n, x, y, device_context);
  }
};

REGISTER_HIP_OPERATOR(
    Sqr,
    UnaryElementwiseOp<TensorTypes<float>, HIPContext, SqrCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    Pow,
    UnaryElementwiseWithArgsOp<TensorTypes<float>, HIPContext, PowFunctor>);
}

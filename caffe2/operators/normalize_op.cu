#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/normalize_op.h"

namespace caffe2 {

__global__ void
NormalizeKernel(const int M, const int N, const float* data_in, float* out) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    float sum_squares = 0.0;
    __shared__ float row_sum_squares;
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const float x_ij = data_in[i * N + j];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum_squares);

    if (threadIdx.x == 0) {
      row_sum_squares = reduce_result;
    }
    __syncthreads();
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
      out[index] = data_in[index] / sqrt(row_sum_squares);
    }
  }
}

__global__ void NormalizeGradientKernel(
    const int M,
    const int N,
    const float* in_mat,
    const float* grad_out_mat,
    float* grad_mat) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage_sum;
  __shared__ BlockReduce::TempStorage temp_storage_norm;
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    float sum = 0.0;
    float norm = 0.0;
    __shared__ float row_sum;
    __shared__ float row_norm;
    __shared__ float row_norm_3;
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
      sum += in_mat[index] * grad_out_mat[index];
      norm += in_mat[index] * in_mat[index];
    }
    float reduce_result = BlockReduce(temp_storage_sum).Sum(sum);
    float reduce_norm = BlockReduce(temp_storage_norm).Sum(norm);

    if (threadIdx.x == 0) {
      row_sum = reduce_result;
      row_norm = sqrt(reduce_norm);
      row_norm_3 = pow(row_norm, 3);
    }
    __syncthreads();
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const int index = i * N + j;
      const float x_ij = in_mat[index];
      const float dy_ij = grad_out_mat[index];
      grad_mat[index] = (dy_ij / row_norm) - ((x_ij / row_norm_3) * row_sum);
    }
  }
}

template <>
bool NormalizeOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  Y->ResizeLike(X);
  int N = X.dim32(X.ndim() - 1);
  int M = X.size() / N;
  NormalizeKernel<<<
      min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      M, N, X.data<float>(), Y->mutable_data<float>());
  return true;
}

template <>
bool NormalizeGradientOp<float, HIPContext>::RunOnDevice() {
  const auto& X = Input(0);
  const auto& dY = Input(1);
  auto* dX = Output(0);
  dX->ResizeLike(X);
  int N = X.dim32(X.ndim() - 1);
  int M = X.size() / N;
  NormalizeGradientKernel<<<
      min(M, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      M,
      N,
      X.data<float>(),
      dY.data<float>(),
      dX->mutable_data<float>());
  return true;
}

namespace {
__global__ void NormalizeL1Kernel(
    const int m,
    const int n,
    const int sf,
    const float* xData,
    float* yData) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;

  for (int i = blockIdx.x; i < n; i += gridDim.x) {
    auto base = (i / sf) * sf * m + (i % sf);

    float sum = 0.0;
    __shared__ float norm;
    for (int j = threadIdx.x; j < m; j += blockDim.x) {
      const auto x_ij = xData[base + j * sf];
      sum += abs(x_ij);
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0) {
      norm = reduce_result;
    }
    __syncthreads();
    if (norm != 0) {
      for (int j = threadIdx.x; j < m; j += blockDim.x) {
        const auto index = base + j * sf;
        yData[index] = xData[index] / norm;
      }
    }
  }
}
} // namespace

template <>
void NormalizeL1Op<float, HIPContext>::DoNormalize(
    const float* xData,
    float* yData,
    const int m,
    const int n,
    const int sf) {
  NormalizeL1Kernel<<<
      min(n, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(m, n, sf, xData, yData);
}

REGISTER_HIP_OPERATOR(Normalize, NormalizeOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    NormalizeGradient,
    NormalizeGradientOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(NormalizeL1, NormalizeL1Op<float, HIPContext>);
} // namespace

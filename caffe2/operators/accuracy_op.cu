#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/accuracy_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {
__global__ void AccuracyKernel(const int N, const int D, const float* Xdata,
    const int* labeldata, float* accuracy) {
  int count = 0;
  CUDA_1D_KERNEL_LOOP(i, N) {
    float maxval = Xdata[i * D];
    int maxid = 0;
    for (int j = 1; j < D; ++j) {
      if (Xdata[i * D + j] > maxval) {
        maxval = Xdata[i * D + j];
        maxid = j;
      }
    }
    if (maxid == labeldata[i]) {
      ++count;
    }
  }
  atomicAdd(accuracy, static_cast<float>(count));
}
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  *accuracy /= N;
}
}  // namespace

template <>
bool AccuracyOp<float, HIPContext>::RunOnDevice() {
  CAFFE_ENFORCE_EQ(
      top_k_, 1, "Currently only top-1 accuracy supported");
  auto& X = Input(PREDICTION);
  auto& label = Input(LABEL);
  auto* Y = Output(0);
  DCHECK_EQ(X.ndim(), 2);
  int N = X.dim32(0);
  int D = X.dim32(1);
  DCHECK_EQ(label.ndim(), 1);
  DCHECK_EQ(label.dim32(0), N);
  Y->Resize(vector<TIndex>());
  float* Ydata = Y->mutable_data<float>();
  math::Set<float, HIPContext>(1, 0, Ydata, &context_);
  AccuracyKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                   0, context_.hip_stream()>>>(
      N, D, X.data<float>(), label.data<int>(), Ydata);
  // This is going to be executed only in one single kernel. Not very beautiful,
  // but probably we have to do this?
  AccuracyDivideKernel<<<1, 1, 0, context_.hip_stream()>>>(
      N, Ydata);
  return true;
}

namespace {
REGISTER_HIP_OPERATOR(Accuracy, AccuracyOp<float, HIPContext>);
}  // namespace
}  // namespace caffe2

#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/boolean_mask_ops.h"

#include <hipcub/hipcub.hpp>

namespace caffe2 {

namespace {
template <typename T>
__global__ void BooleanMaskCopyKernel(
    const TIndex numOfOutput,
    const TIndex numBytes,
    const TIndex* indices,
    const T* src,
    T* dest) {
  for (TIndex i = blockIdx.x; i < numOfOutput; i += gridDim.x) {
    const auto srcBase = indices[i] * numBytes;
    const auto destBase = i * numBytes;
    for (TIndex j = threadIdx.x; j < numBytes; j += blockDim.x) {
      dest[destBase + j] = src[srcBase + j];
    }
  }
}
}

template <>
class BooleanMaskOp<HIPContext> final : public Operator<HIPContext> {
 public:
  BooleanMaskOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws) {}

  bool RunOnDevice() override {
    const auto& src = Input(0);
    const auto& mask = Input(1);
    auto* dest = Output(0);

    CAFFE_ENFORCE(src.ndim() >= 1);
    CAFFE_ENFORCE_EQ(mask.ndim(), 1);
    CAFFE_ENFORCE(src.dims()[0] == mask.dims()[0]);

    const auto* maskData = mask.data<bool>();
    const auto outerSize = mask.dims()[0];
    indices_.Resize(outerSize);
    auto* indicesData = indices_.mutable_data<TIndex>();

    size_t numBytes = 0;
    hipcub::CountingInputIterator<int> itr(0);
    hipcub::DeviceSelect::Flagged(
        nullptr,
        numBytes,
        itr,
        maskData,
        indicesData,
        static_cast<TIndex*>(nullptr),
        outerSize,
        context_.hip_stream());

    auto numTIndex =
        static_cast<TIndex>((numBytes + sizeof(TIndex) - 1) / sizeof(TIndex));
    // allocate one more TIndex at the end of scratch for storing numOfOutput
    scratch_.Resize(numTIndex + 1);
    auto* scratchData = scratch_.mutable_data<TIndex>();
    auto* numOfOutputData = scratchData + numTIndex;

    hipcub::DeviceSelect::Flagged(
        static_cast<void*>(scratchData),
        numBytes,
        itr,
        maskData,
        indicesData,
        numOfOutputData,
        outerSize,
        context_.hip_stream());

    // Copy numOfOutput from gpu to cpu
    TIndex numOfOutput;
    context_.Copy<TIndex, HIPContext, CPUContext>(
        1, numOfOutputData, &numOfOutput);

    indices_.Resize(numOfOutput);
    std::vector<TIndex> dims = src.dims();
    dims[0] = numOfOutput;
    dest->Resize(dims);
    auto* destData = (char*)dest->raw_mutable_data(src.meta());
    const auto* srcData = (char*)src.raw_data();
    if (OutputSize() == 2) {
      auto* indicesOut = Output(1);
      indicesOut->Resize(numOfOutput);
      indicesOut->mutable_data<TIndex>();
    }

    if (numOfOutput > 0) {
      BooleanMaskCopyKernel<<<
          min(numOfOutput, static_cast<TIndex>(CAFFE_MAXIMUM_NUM_BLOCKS)),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          numOfOutput,
          src.size_from_dim(1) * src.meta().itemsize(),
          indicesData,
          srcData,
          destData);

      if (OutputSize() == 2) {
        Output(1)->CopyFrom(indices_, &context_);
      }
    }

    return true;
  }

 private:
  Tensor<HIPContext> indices_;
  Tensor<HIPContext> scratch_;
};

REGISTER_HIP_OPERATOR(BooleanMask, BooleanMaskOp<HIPContext>);

namespace {

#define minf (-1.0f * std::numeric_limits<float>::infinity())

__global__ void sequenceMaskKernel(
    int N,
    int D,
    const float* in,
    const int* seq_lengths,
    float fill_val,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] = (j >= seq_lengths[i] ? fill_val : in[index]);
  }
}

__global__ void windowMaskKernel(
    int N,
    int D,
    const float* in,
    const int* window_centers,
    const int radius,
    float fill_val,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] =
        (j < window_centers[i] - radius || j > window_centers[i] + radius
             ? fill_val
             : in[index]);
  }
}

__global__ void
upperMaskKernel(int N, int D, const float* in, float fill_val, float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] = (j > i ? fill_val : in[index]);
  }
}

__global__ void
lowerMaskKernel(int N, int D, const float* in, float fill_val, float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] = (j < i ? fill_val : in[index]);
  }
}

__global__ void
upperDiagMaskKernel(int N, int D, const float* in, float fill_val, float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] = (j >= i ? fill_val : in[index]);
  }
}

__global__ void
lowerDiagMaskKernel(int N, int D, const float* in, float fill_val, float* out) {
  CUDA_1D_KERNEL_LOOP(index, N * D) {
    int i = index / D;
    int j = index % D;

    out[index] = (j <= i ? fill_val : in[index]);
  }
}

} // namespace

template <>
bool SequenceMaskOp<HIPContext>::RunOnDevice() {
  const Tensor<HIPContext>* input = &Input(0);
  const Tensor<HIPContext>* sequence_lengths = nullptr;
  const Tensor<HIPContext>* window_centers = nullptr;

  if (mode_ == "sequence") {
    sequence_lengths = &Input(1);
  } else if (mode_ == "window") {
    window_centers = &Input(1);
  }

  auto* output = Output(0);
  output->ResizeLike(*input);

  const auto canonical_axis = input->canonical_axis_index(axis_);
  const int left = input->size_to_dim(canonical_axis);
  const int right = input->size_from_dim(canonical_axis);

  float fill_val = (grad_ ? 0.0f : fill_val_);

  if (mode_ == "sequence") {
    sequenceMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        sequence_lengths->data<int>(),
        fill_val,
        output->mutable_data<float>());
  } else if (mode_ == "window") {
    windowMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        window_centers->data<int>(),
        radius_,
        fill_val,
        output->mutable_data<float>());
  } else if (mode_ == "upper") {
    upperMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        fill_val,
        output->mutable_data<float>());
  } else if (mode_ == "lower") {
    lowerMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        fill_val,
        output->mutable_data<float>());
  } else if (mode_ == "upperdiag") {
    upperDiagMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        fill_val,
        output->mutable_data<float>());
  } else if (mode_ == "lowerdiag") {
    lowerDiagMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        input->data<float>(),
        fill_val,
        output->mutable_data<float>());
  } else {
    CAFFE_ENFORCE(false, "Unsupported mode for SequenceMaskOp!");
  }

  return true;
}

REGISTER_HIP_OPERATOR(SequenceMask, SequenceMaskOp<HIPContext>);

} // caffe2

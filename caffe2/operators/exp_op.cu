#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void ExpKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __expf(X[i]);
  }
}

struct ExpCUDAFunctor {
  template <typename T>
  inline void operator()(const int n, const T* x,
                         T* y, HIPContext* device_context) {
    ExpKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
  inline bool InplaceAllowed() {
    return true;
  }
};

REGISTER_HIP_OPERATOR(
    Exp, UnaryElementwiseOp<TensorTypes<float>, HIPContext, ExpCUDAFunctor>);
}  // namespace caffe2

#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/softplus_op.h"

namespace caffe2 {
namespace {
template <typename T>
__global__ void SoftplusKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = log(exp(X[i]) + 1.0f);
  }
}

template <typename T>
__global__ void
SoftplusGradientKernel(const int N, const T* Y, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float nexpY = exp(-Y[i]);
    dX[i] = dY[i] * (1 - nexpY);
  }
}
} // namespace

template <>
bool SoftplusOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  DCHECK_GT(X.size(), 0);
  Y->ResizeLike(X);
  SoftplusKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(), X.data<float>(), Y->mutable_data<float>());
  return true;
}

template <>
bool SoftplusGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);
  auto* dX = Output(0);
  DCHECK_GT(Y.size(), 0);
  DCHECK_EQ(dY.size(), Y.size());
  dX->ResizeLike(Y);
  SoftplusGradientKernel<<<
      CAFFE_GET_BLOCKS(Y.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      Y.size(), Y.data<float>(), dY.data<float>(), dX->mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Softplus, SoftplusOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SoftplusGradient,
    SoftplusGradientOp<float, HIPContext>);
} // namespace caffe2

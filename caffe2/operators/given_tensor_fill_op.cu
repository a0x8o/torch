#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/given_tensor_fill_op.h"

namespace caffe2 {

REGISTER_HIP_OPERATOR(GivenTensorFill, GivenTensorFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(GivenTensorIntFill, GivenTensorFillOp<int, HIPContext>);
REGISTER_HIP_OPERATOR(
    GivenTensorBoolFill,
    GivenTensorFillOp<bool, HIPContext>);
}

#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_op.h"

namespace caffe2 {

template <typename T>
__global__ void TanhKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = tanh(X[i]);
  }
}

template <typename T>
__global__ void TanhGradientKernel(const int N, const T* Y, const T* dY,
                              T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = dY[i]*(1 - Y[i]*Y[i]);
  }
}

struct TanhCUDAFunctor {
  template <typename T>
  inline void operator()(const int n, const T* x,
                         T* y, HIPContext* device_context) {
    TanhKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                    0, device_context->hip_stream()>>>(n, x, y);
    return;
  }
  inline bool InplaceAllowed() {
    return true;
  }
};

struct TanhGradientCUDAFunctor {
  template <typename T>
  inline void Run(const int n, const T* y, const T* dy,
                  T* dx, HIPContext* device_context) {
    TanhGradientKernel<T><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                            0, device_context->hip_stream()>>>(n, y, dy, dx);
    return;
  }
};

REGISTER_HIP_OPERATOR(
    Tanh, UnaryElementwiseOp<TensorTypes<float>, HIPContext, TanhCUDAFunctor>);
REGISTER_HIP_OPERATOR(
    TanhGradient, BinaryElementwiseOp<
        TensorTypes<float>, HIPContext,
        WithoutBroadcast<TanhGradientCUDAFunctor>>);
}  // namespace caffe2

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/loss_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(AveragedLoss, AveragedLoss<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    AveragedLossGradient,
    AveragedLossGradient<float, HIPContext>);
}  // namespace
}  // namespace caffe2

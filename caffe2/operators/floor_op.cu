#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/floor_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

template <typename T>
__global__ void FloorKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = std::floor(X[i]);
  }
}

template <>
bool FloorOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  CAFFE_ENFORCE_GT(X.size(), 0);
  Y->ResizeLike(X);
  FloorKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(), X.data<float>(), Y->mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Floor, FloorOp<float, HIPContext>);

} // namespace caffe2

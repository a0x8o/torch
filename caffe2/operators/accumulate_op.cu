#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/accumulate_op.h"

namespace caffe2 {
namespace {
REGISTER_HIP_OPERATOR(Accumulate, AccumulateOp<float, HIPContext>);
}  // namespace
}  // namespace caffe2

#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/filler_op.h"

namespace caffe2 {

namespace {
__global__ void FillRangeKernel(const int n, float* data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    data[index] = index;
  }
}
}

template <>
bool RangeFillOp<float, HIPContext>::Fill(
    TensorCUDA* output) {
  int N = output->size();
  FillRangeKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                    0, context_.hip_stream()>>>(
      N, output->mutable_data<float>());
  return true;
}

namespace {

REGISTER_HIP_OPERATOR(UniformFill, UniformFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(UniformIntFill, UniformFillOp<int, HIPContext>);
REGISTER_HIP_OPERATOR(ConstantFill, ConstantFillOp<HIPContext>);
REGISTER_HIP_OPERATOR(GaussianFill, GaussianFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(XavierFill, XavierFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MSRAFill, MSRAFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(RangeFill, RangeFillOp<float, HIPContext>);

}  // namespace
}  // namespace caffe2

#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "caffe2/operators/order_switch_ops.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void NHWC2NCHWKernel(const int N, const int HW, const int C,
                                const float* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N * HW * C) {
    const int c = i % C;
    const int hw = i / C % HW;
    const int n = i / C / HW;
    Y[(n * C + c) * HW + hw] = X[i];
  }
}

__global__ void NCHW2NHWCKernel(const int N, const int C, const int HW,
                                const float* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N * C * HW) {
    const int hw = i % HW;
    const int c = i / HW % C;
    const int n = i / C / HW;
    Y[(n * HW + hw) * C + c] = X[i];
  }
}

template <>
bool NHWC2NCHWOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  DCHECK_EQ(X.ndim(), 4);
  const int N = X.dim32(0), H = X.dim32(1), W = X.dim32(2), C = X.dim32(3);
  Y->Resize(N, C, H, W);
  NHWC2NCHWKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                    0, context_.hip_stream()>>>(
      N, H * W, C, X.data<float>(), Y->mutable_data<float>());
  return true;
}

template <>
bool NCHW2NHWCOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  DCHECK_EQ(X.ndim(), 4);
  const int N = X.dim32(0), C = X.dim32(1), H = X.dim32(2), W = X.dim32(3);
  Y->Resize(N, H, W, C);
  NCHW2NHWCKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                    0, context_.hip_stream()>>>(
      N, C, H * W, X.data<float>(), Y->mutable_data<float>());
  return true;
}


REGISTER_HIP_OPERATOR(NHWC2NCHW, NHWC2NCHWOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(NCHW2NHWC, NCHW2NHWCOp<float, HIPContext>);
}  // namespace caffe2

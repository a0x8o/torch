#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rmsprop_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void RmsPropUpdate(
    int N,
    const float* g,
    const float* ms,
    const float* mom,
    float* ng,
    float* nms,
    float* nmom,
    float decay,
    float momentum,
    float epsilon,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Update new mean square estimate
    nms[i] = ms[i] + (1.0f - decay) * (g[i] * g[i] - ms[i]);
    // Update momentum estimate
    nmom[i] =
        mom[i] * momentum + lr[0] * g[i] / std::sqrt(epsilon + nms[i]);
    // New gradient is the momentum
    ng[i] = nmom[i];
  }
}

template <>
void rmsprop_update<HIPContext>(
    int N,
    const float* g,
    const float* ms,
    const float* mom,
    float* ng,
    float* nms,
    float* nmom,
    float decay,
    float momentum,
    float epsilon,
    const float* lr,
    HIPContext* context) {
  RmsPropUpdate<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
      N, g, ms, mom, ng, nms, nmom, decay, momentum, epsilon, lr);
}


REGISTER_HIP_OPERATOR(RmsProp, RmsPropOp<float, HIPContext>);

}
